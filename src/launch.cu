#include "easyprof.h"

#include <cxxabi.h>
#include <dlfcn.h>
#include <functional>

#define GPU_FUNC_LAUNCH_BEGIN(prefix, __stream, __function, \
	RetTy, name, ...) \
	extern "C" \
	RetTy api_name(name, prefix)(__VA_ARGS__) \
	{ \
		gpuStream_t __s = static_cast<gpuStream_t>(__stream); \
		return gpuFuncLaunch<RetTy>( \
			__dll(str_prefix(prefix)), str_api_name(name, prefix), __s, __function,

#define GPU_FUNC_LAUNCH_END(...) \
			__VA_ARGS__); \
	}

// CUDA/HIP APIs can execute a user callback function, when the corresponding
// stream reaches the point of interest. We use this feature to track kernels
// execution in a simple way.
#ifdef __HIPCC__
static void profilerTimerSync(hipStream_t stream, hipError_t status, void *userData)
#else
static void profilerTimerSync(hipStream_t stream, hipError_t status, void *userData)
#endif
{
	if (Profiler::get().timer->isTiming())
		Profiler::get().timer->sync(stream);
}

// This is a reverse-engineering of some internal CUDA structures,
// in order to reach out some data, most importantly to the kernel name.

struct kernel
{
	uint32_t v0;
	uint32_t v1;
	uint32_t v2;
	uint64_t v3;
	uint32_t v4;
	uint32_t v5;
	uint32_t v6;
	uint32_t v7;
	uint32_t v8;
	void *module;
	uint32_t size;
	uint32_t v9;
	void *p1;   
};

struct dummy1
{
	void *p0;
	void *p1;
	uint64_t v0;
	uint64_t v1;
	void *p2;
};

struct ihipModuleSymbol_t
{
	uint32_t v0;
	uint32_t v1;
	char *name;
	uint32_t v2;
	uint32_t v3;
	uint32_t v4;
	uint32_t v5;
	struct kernel *kernel;
	void *p1;
	void *p2;
	uint32_t v6;
	uint32_t v7;
	uint32_t v8;
	uint32_t v9;
	uint32_t v10;
	uint32_t v11;
	uint32_t v12;
	uint32_t v13;
	uint32_t v14;
	uint32_t v15;
	uint32_t v16;
	uint32_t v17;
	uint32_t v18;
	uint32_t v19;
	uint32_t v20;
	uint32_t v21;
	uint32_t v22;
	uint32_t v23;
	struct dummy1 *p3;
};

template<typename RetTy, typename Function, typename... Args>
RetTy gpuFuncLaunch(const std::string dll, const std::string sym, gpuStream_t stream, Function f,
	unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ,
	unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ,
	unsigned int sharedMemBytes, Args... args)
{
	void* handle = nullptr;
	{
		auto it = dlls.find(dll);
		if (it != dlls.end())
			handle = it->second;
		else
		{
			handle = dlopen(dll.c_str(), RTLD_NOW | RTLD_GLOBAL);
			if (!handle)
			{
				LOG("Error loading %s: %s", dll.c_str(), dlerror());
				abort();
			}
			dlls.insert(std::make_pair(dll, handle));
		}
	}

	using Func = RetTy (*)(Args...);
	
	static Func funcReal = nullptr;
	if (!funcReal)
	{
		funcReal = (Func)SymbolLoader::get(handle, sym.c_str());
		if (!funcReal)
		{
			LOG("Error loading %s : %s", sym.c_str(), dlerror());
			abort();
		}
	}

	auto it = Profiler::get().funcs.find(reinterpret_cast<const void*>(f));
	if (it == Profiler::get().funcs.end())
	{
		struct ihipModuleSymbol_t *pFunc = (struct ihipModuleSymbol_t *)f;
		struct kernel *pKernel = pFunc->kernel;
#ifdef __HIPCC__
		int status;    
		char* name = abi::__cxa_demangle(pFunc->name, 0, 0, &status);	
		auto deviceName = status ? pFunc->name : name;
#else
		const char* deviceName = "(unknown)";
#endif
		// Get the kernel register count.
		int nregs = 0;
#ifdef __HIPCC__
		if (hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, pFunc) != hipSuccess)
		{
			fprintf(stderr, "Could not read the number of registers for function \"%s\"\n", deviceName);
			auto err = gpuGetLastError();
		}
#else
		struct gpuFuncAttributes attrs;
		if (gpuFuncGetAttributes(&attrs, (void*)f) != gpuSuccess)
		{
			fprintf(stderr, "Could not read the number of registers for function \"%s\"\n", deviceName);
			auto err = gpuGetLastError();
		}
		else
		{
			nregs = attrs.numRegs;
		}
#endif
		auto result = Profiler::get().funcs.emplace(reinterpret_cast<const void*>(f),
			std::make_shared<GPUfunction>(GPUfunction
		{
			/* std::string deviceName; */      deviceName,
			/* char* deviceFun; */             f,
#ifdef __HIPCC__
			/* void* module */                 pKernel->module,
#else
			/* void* module */                 nullptr,
#endif
			/* unsigned int sharedMemBytes; */ sharedMemBytes,
			/* int nregs; */                   nregs
		}));

		it = result.first;
	}

	// Call the real function.
	auto result = std::invoke(funcReal, args...);

	auto& func = it->second;
	auto& name = func->deviceName;

	// Start profiling the newly-launched kernel.
	if (Profiler::get().matcher->isMatching(name))
	{
		// Don't do anything else, if kernel launch was not successful.
		if (result != hipSuccess) return result;
		
		if (Profiler::get().timer->isTiming())
		{
			Profiler::get().timer->measure(func.get(),
				dim3(gridDimX, gridDimY, gridDimZ),
				dim3(blockDimX, blockDimY, blockDimZ),
				stream);
#ifdef __HIPCC__			
			// Insert a callback into the same stream after the launch,
			// in order to have it to stop the time measurement.
			auto err = hipStreamAddCallback(stream, profilerTimerSync, /* userData = */ nullptr, 0);
#else
			// in order to have it to stop the time measurement.
			auto err = hipStreamAddCallback(stream, profilerTimerSync, /* userData = */ nullptr, 0);
#endif
		}
	}

	return result;
}

#if defined(__HIPCC__)

// HIP has multiple different API functions for kernel launching.

GPU_FUNC_LAUNCH_BEGIN(RuntimeLibraryPrefix, stream, f,
	gpuError_t, LaunchKernel,
	const void* f, dim3 numBlocks, dim3 dimBlocks, void** args, size_t sharedMemBytes, gpuStream_t stream)
GPU_FUNC_LAUNCH_END(numBlocks.x, numBlocks.y, numBlocks.z,
	dimBlocks.x, dimBlocks.y, dimBlocks.z, sharedMemBytes,
	f, numBlocks, dimBlocks, args, sharedMemBytes, stream);

GPU_FUNC_LAUNCH_BEGIN(RuntimeLibraryPrefix, stream, f,
	gpuError_t, ExtLaunchKernel,
	const void* f, dim3 numBlocks, dim3 dimBlocks, void** args, size_t sharedMemBytes, gpuStream_t stream,
	gpuEvent_t startEvent, gpuEvent_t stopEvent, int flags)
GPU_FUNC_LAUNCH_END(numBlocks.x, numBlocks.y, numBlocks.z,
	dimBlocks.x, dimBlocks.y, dimBlocks.z, sharedMemBytes,
	f, numBlocks, dimBlocks, args, sharedMemBytes, stream, startEvent, stopEvent, flags);

GPU_FUNC_LAUNCH_BEGIN(RuntimeLibraryPrefix, stream, f,
	gpuError_t, ModuleLaunchKernel,
	gpuFunction_t f, unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ,
	unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ, unsigned int sharedMemBytes,
	gpuStream_t stream, void** kernelParams, void** extra)
GPU_FUNC_LAUNCH_END(gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes,
	f, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes,
	stream, kernelParams, extra);

#else

// CUDA has Runtime API and Driver API functions for kernel launching,
// but the former in turn calls the latter, so we need to handle the driver API only.

GPU_FUNC_LAUNCH_BEGIN(DriverLibraryPrefix, hStream, f,
	hipError_t, LaunchKernel,
	hipFunction_t f,
	unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ,
	unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ,
	unsigned int sharedMemBytes, hipStream_t hStream, void** kernelParams, void** extra)
GPU_FUNC_LAUNCH_END(gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes,
	f, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, hStream, kernelParams, extra);

#endif

