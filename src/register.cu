#include "hip/hip_runtime.h"
#include "easyprof.h"

#include <cxxabi.h>
#include <dlfcn.h>

#define bind_lib(path, lib) \
if (!lib) \
{ \
	lib = dlopen(path, RTLD_NOW | RTLD_GLOBAL); \
	if (!lib) \
	{ \
		LOG("Error loading %s: %s", path, dlerror()); \
		abort(); \
	} \
}

static std::string old_api_name(const std::string& sym)
{
	std::string result = sym;
#if __HIPCC__
	return result.replace(result.find("gpu"), 3, "hip");
#else
	return result.replace(result.find("gpu"), 3, "cuda");
#endif
}

#define bind_sym(handle, sym, retty, ...) \
typedef retty (*sym##_func_t)(__VA_ARGS__); \
static sym##_func_t sym##_real = nullptr; \
if (!sym##_real) \
{ \
	auto name = old_api_name(#sym); \
	sym##_real = (sym##_func_t)SymbolLoader::get(handle, name.c_str()); \
	if (!sym##_real) \
	{ \
		LOG("Error loading %s : %s", name.c_str(), dlerror()); \
		abort(); \
	} \
}

extern "C"
void __gpuRegisterFunction(
	void** vfatCubinHandle,
	const char* hostFun,
	char* deviceFun,
	const char* deviceName,
	int thread_limit,
	uint3* tid,
	uint3* bid,
	dim3* bDim,
	dim3* gDim,
	int* wSize)
{
	bind_lib(LIBGPURT, libgpurt);
	bind_sym(libgpurt, __gpuRegisterFunction, void,
		void**, const char*, char*, const char*,
		int, uint3*, uint3*, dim3*, dim3*, int*);

	__gpuRegisterFunction_real(
		vfatCubinHandle, hostFun, deviceFun, deviceName,
		thread_limit, tid, bid, bDim, gDim, wSize);
#if 0
#define VAL_OR_NIL(ptr, prop) (ptr ? ((ptr)->prop) : 0)
	LOG("__gpuRegisterFunction(\"%s\", %p, %p, %p, %d, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %d)\n",
		deviceName, vfatCubinHandle, hostFun, deviceFun, thread_limit,
		VAL_OR_NIL(tid, x), VAL_OR_NIL(tid, y), VAL_OR_NIL(tid, z),
		VAL_OR_NIL(bid, x), VAL_OR_NIL(bid, y), VAL_OR_NIL(bid, z),
		VAL_OR_NIL(bDim, x), VAL_OR_NIL(bDim, y), VAL_OR_NIL(bDim, z),
		VAL_OR_NIL(gDim, x), VAL_OR_NIL(gDim, y), VAL_OR_NIL(gDim, z),
		wSize ? *wSize : 0);
#endif
	int status;    
	char* name = abi::__cxa_demangle(deviceName, 0, 0, &status);

	auto uint3zero = uint3{};
	auto dim3zero = dim3{};
	profiler.funcs[(void*)hostFun] = std::make_shared<GPUfunction>(GPUfunction
	{
		vfatCubinHandle,
		hostFun,
		deviceFun,
		status ? deviceName : name,
		thread_limit,
		(tid ? *tid : uint3zero),
		(bid ? *bid : uint3zero),
		(bDim ? *bDim : dim3zero),
		(gDim ? *gDim : dim3zero),
		(wSize ? *wSize : 0),
		0 // nregs, not available yet
	});
}

