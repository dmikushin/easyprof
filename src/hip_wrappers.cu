
#include <hip/hip_runtime.h>
#ifdef __HIPCC__

#include "easyprof.h"

std::vector<GPUapiFunc> gpuAPI =
{
	GPU_API_NAME(hipLaunchKernel),
	GPU_API_NAME(hipExtLaunchKernel),
	GPU_API_NAME(hipModuleLaunchKernel)
};

#endif // __CUDACC__
