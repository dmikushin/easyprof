#ifdef __HIPCC__

#include "easyprof.h"

std::vector<GPUapiFunc> gpuAPI =
{
	GPU_API_NAME(hipGetProcAddress),

	GPU_API_NAME(hipModuleLaunchCooperativeKernel),
	GPU_API_NAME(hipLaunchHostFunc),
	GPU_API_NAME(hipModuleLaunchKernel)
};

#endif // __HIPCC__

