#ifdef __HIPCC__

#include "easyprof.h"

std::vector<GPUapiFunc> gpuAPI =
{
	GPU_API_NAME(hipGetProcAddress),

	GPU_API_NAME(hipMalloc),
	GPU_API_NAME(hipMallocFromPoolAsync),
	GPU_API_NAME(hipMemAllocHost),
	GPU_API_NAME(hipMallocManaged),
	GPU_API_NAME(hipMemAllocPitch),

	GPU_API_NAME(cuMemcpy),
	GPU_API_NAME(hipMemcpyParam2D),
	GPU_API_NAME(hipMemcpyParam2DAsync),
	GPU_API_NAME(hipDrvMemcpy2DUnaligned),
	GPU_API_NAME(hipDrvMemcpy3D),
	GPU_API_NAME(hipDrvMemcpy3DAsync),
	GPU_API_NAME(cuMemcpy3DPeer),
	GPU_API_NAME(cuMemcpy3DPeerAsync),
	GPU_API_NAME(cuMemcpyAsync),
	GPU_API_NAME(hipMemcpyAtoA),
	GPU_API_NAME(hipMemcpyAtoD),
	GPU_API_NAME(hipMemcpyAtoH),
	GPU_API_NAME(hipMemcpyAtoHAsync),
	GPU_API_NAME(hipMemcpyDtoA),
	GPU_API_NAME(hipMemcpyDtoD),
	GPU_API_NAME(hipMemcpyDtoDAsync),
	GPU_API_NAME(hipMemcpyDtoH),
	GPU_API_NAME(hipMemcpyDtoHAsync),
	GPU_API_NAME(hipMemcpyHtoA),
	GPU_API_NAME(hipMemcpyHtoAAsync),
	GPU_API_NAME(hipMemcpyHtoD),
	GPU_API_NAME(hipMemcpyHtoDAsync),
	GPU_API_NAME(cuMemcpyPeer),
	GPU_API_NAME(cuMemcpyPeerAsync),

	GPU_API_NAME(hipMemsetD16),
	GPU_API_NAME(hipMemsetD16Async),
	GPU_API_NAME(hipMemsetD2D16),
	GPU_API_NAME(hipMemsetD2D16Async),
	GPU_API_NAME(hipMemsetD2D32),
	GPU_API_NAME(hipMemsetD2D32Async),
	GPU_API_NAME(hipMemsetD2D8),
	GPU_API_NAME(hipMemsetD2D8Async),
	GPU_API_NAME(hipMemsetD32),
	GPU_API_NAME(hipMemsetD32Async),
	GPU_API_NAME(hipMemsetD8),
	GPU_API_NAME(hipMemsetD8Async),

	GPU_API_NAME(hipModuleLaunchCooperativeKernel),
	GPU_API_NAME(hipLaunchHostFunc),
	GPU_API_NAME(hipModuleLaunchKernel)
};

#endif // __HIPCC__

